#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>
#include <memory>


//������������
__constant__ int constant_f;
__constant__ int constant_g;
#define N 5


//�ں˺���Ϊ��ʹ�ó����ڴ�
__global__ void gpu_constant_memory(float* d_in, float* d_out) {
	//������ǰ�ں˵��߳�����
	int tid = threadIdx.x;
	d_out[tid] = constant_f * d_in[tid] + constant_g;
}

int main(void) {
	//Ϊ������������
	float h_in[N], h_out[N];
	//Ϊ�豸����ָ��
	float* d_in, * d_out;
	int h_f = 2;
	int h_g = 20;

	//��cpu�Ϸ����ڴ�
	hipMalloc((void**)&d_in, N * sizeof(float));
	hipMalloc((void**)&d_out, N * sizeof(float));

	//��ʼ������
	for (int i = 0; i < N; i++)
	{
		h_in[i] = i;
	}
	//�������������鵽�豸
	hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
	//���Ƴ����������ڴ�
	hipMemcpyToSymbol(HIP_SYMBOL(constant_f), &h_f, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(constant_g), &h_g, sizeof(int), 0, hipMemcpyHostToDevice);

	//�����ں˴���һ�����N���߳�ÿ����
	gpu_constant_memory << <1, N >> > (d_in, d_out);

	//���豸�ڴ渴�ƽ�����ظ�����
	hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

	//��ӡ����������
	printf("Use of Constant memory on GPU\n");
	for (int i = 0; i < N; i++)
	{
		printf("The expression for index %f is %f\n", h_in[i], h_out[i]);
	}
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}
